#include <stdio.h>
#include <stdlib.h>
#include <string>
#include <cstring>
#include <vector>
#include <time.h>
#include <sstream>
#include <iostream>
#include <fstream>
#include <cmath>
#include <numeric>
#include <hip/hip_runtime.h>
#include <utility>
//#include "helper_cuda.h"

using namespace std;


// Función para multiplicar matrices cuadradas en CUDA.
__global__ void matrixMul(const float *A, const float *B, float *C, int numRows)
{
    int row = blockIdx.y * blockDim.y + threadIdx.y;
    int col = blockIdx.x * blockDim.x + threadIdx.x;

    int numCols = numRows;

    if (row < numRows and col < numCols){
        int aux = 0;
        for (int i = 0; i < numRows; ++i){
          aux += A[row * numCols + i] * B[col + numRows * i];
        }

        C[row * numCols + col] = aux;
    }
}

/**
 * Función de cálculo de tiempos.
 * Usa clock_gettime.
 */
timespec diff(timespec start, timespec end){
     timespec temp;
     if ((end.tv_nsec - start.tv_nsec) < 0) {
       temp.tv_sec = end.tv_sec - start.tv_sec - 1;
       temp.tv_nsec = 1000000000 + end.tv_nsec - start.tv_nsec;
     }
     else{
       temp.tv_sec = end.tv_sec - start.tv_sec;
       temp.tv_nsec = end.tv_nsec - start.tv_nsec;
     }
     return temp;
}

/**
 * Función de lectura de archivos.
 * Lee una matriz de un archivo y la devuelve.
 * Trabajamos con la matriz como un array.
 */
float* readMatrix(string file){
  ifstream ifs;
  ifs.open(file, ifstream::in);

  // Leemos elementos
  int rows, cols;
  ifs >> rows;
  ifs >> cols;

  float* resultado = (float *)malloc(rows * cols *sizeof(float));

  // Lectura.
  for (int i = 0; i < rows; ++i)
    for (int j = 0; j < cols; ++j)
      ifs >> resultado[i * cols + j];

  ifs.close();

  return resultado;
}

// Devuelve las filas y columnas de una matriz en forma de par.
pair<int, int> size(string file){
  ifstream ifs;
  const char* c = file.c_str();
  ifs.open(c, ifstream::in);

  // Leemos tamaño
  int rows, cols;
  ifs >> rows;
  ifs >> cols;

  ifs.close();

  return make_pair(rows, cols);
}
// Manejo de errores
void cudaErrorHandler(hipError_t e, const int E)
{
    if (e != hipSuccess) {
        fprintf(stderr, "ERROR(%d): %s\n", E, hipGetErrorString(e));
        exit(EXIT_FAILURE);
    }
}


int main(int argc, char *argv[])
{

    if (argc != 4){
      cerr << "Necesito tres argumentos, los dos ficheros que contienen las matrices y el nombre de la matriz de salida.\n";
      exit(-1);
    }

    // Error code to check return values for CUDA calls
    hipError_t err = hipSuccess;

    // Datos del programa
    string file1, file2, file3;

    // Lectura de argumentos
    file1 = argv[1];
    file2 = argv[2];
    file3 = argv[3];

    // Para medir el tiempo de suma
    timespec start, finish, dif, t_alloc, t_comp;

    cerr << "Leyendo archivos" << endl;

    // Declaración de matrices.
    // Las consideramos como un array.
    float* m1 = readMatrix(file1);
    float* m2 = readMatrix(file2);

    cerr << "Matrices leídas" << endl;

    pair<int, int> size1 = size(file1);
    pair<int, int> size2 = size(file2);

    if (size1.second != size2.first){
      cerr << "No puedo multiplicar las matrices.\n";
      exit(-1);
    }

    // Tamaño
    size_t size = size1.first * size1.second * sizeof(float);
    // Declaración del vector suma.
    float* m3 = (float *)malloc(size);

    // Inicialización del tiempo
    clock_gettime(CLOCK_REALTIME, &start); 

    cerr << "Empezamos hipMalloc" << endl;

    // Reservaos espacio para las matrices.
    float *d_A = NULL;
    err = hipMalloc((void **)&d_A, size);

    if (err != hipSuccess)
    {
        fprintf(stderr, "Failed to allocate device vector A (error code %s)!\n", hipGetErrorString(err));
        exit(EXIT_FAILURE);
    }

    float *d_B = NULL;
    err = hipMalloc((void **)&d_B, size);


    if (err != hipSuccess)
    {
        fprintf(stderr, "Failed to allocate device vector B (error code %s)!\n", hipGetErrorString(err));
        exit(EXIT_FAILURE);
    }

    float *d_C = NULL;
    err = hipMalloc((void **)&d_C, size);

    if (err != hipSuccess)
    {
        fprintf(stderr, "Failed to allocate device vector C (error code %s)!\n", hipGetErrorString(err));
        exit(EXIT_FAILURE);
    }

    // Copiamos las matrices.
    cerr << "Copia a memoria GPU" << endl;

    err = hipMemcpy(d_A, m1, size, hipMemcpyHostToDevice);

    if (err != hipSuccess)
    {
        fprintf(stderr, "Failed to copy vector A from host to device (error code %s)!\n", hipGetErrorString(err));
        exit(-1);
    }

    err = hipMemcpy(d_B, m2, size, hipMemcpyHostToDevice);

    if (err != hipSuccess)
    {
        fprintf(stderr, "Failed to copy vector B from host to device (error code %s)!\n", hipGetErrorString(err));
        exit(-1);
    }

    clock_gettime(CLOCK_REALTIME, &finish); 
    t_alloc = diff(start, finish); 

    // Producto
    cerr << "Lanzamos producto" << endl;

    clock_gettime(CLOCK_REALTIME, &start); 

    // Launch the Vector Add CUDA Kernel
    int threadsPerBlock = 256;
    int blocksPerGrid =(size1.first + threadsPerBlock - 1) / threadsPerBlock;
    printf("CUDA kernel launch with %d blocks of %d threads\n", blocksPerGrid, threadsPerBlock);
    matrixMul<<<blocksPerGrid, threadsPerBlock>>>(d_A, d_B, d_C, size2.first);
    err = hipGetLastError();

    if (err != hipSuccess)
    {
        fprintf(stderr, "Failed to launch matrixMul kernel (error code %s)!\n", hipGetErrorString(err));
        exit(EXIT_FAILURE);
    }
 
    clock_gettime(CLOCK_REALTIME, &finish); 
    t_comp = diff(start, finish); 

    // Copia en memoria principal  

    cerr << "Copia en memoria principal" << endl;

    // Copy the device result vector in device memory to the host result vector
    // in host memory.
    
    err = hipMemcpy(m3, d_C, size, hipMemcpyDeviceToHost);

    if (err != hipSuccess)
    {
        fprintf(stderr, "Failed to copy vector C from device to host (error code %s)!\n", hipGetErrorString(err));
        exit(EXIT_FAILURE);
    }

    // Finalización del tiempo
    clock_gettime(CLOCK_REALTIME, &finish);
    // Resultado
    dif = diff(start, finish);
    printf("%ld.%09ld, %ld.%09ld, %ld.%09ld\n",
            t_alloc.tv_sec, t_alloc.tv_nsec, t_comp.tv_sec, t_comp.tv_nsec, dif.tv_sec, dif.tv_nsec);


    // Free device global memory
    err = hipFree(d_A);

    if (err != hipSuccess)
    {
        fprintf(stderr, "Failed to free device vector A (error code %s)!\n", hipGetErrorString(err));
        exit(EXIT_FAILURE);
    }

    err = hipFree(d_B);

    if (err != hipSuccess)
    {
        fprintf(stderr, "Failed to free device vector B (error code %s)!\n", hipGetErrorString(err));
        exit(EXIT_FAILURE);
    }

    err = hipFree(d_C);

    if (err != hipSuccess)
    {
        fprintf(stderr, "Failed to free device vector C (error code %s)!\n", hipGetErrorString(err));
        exit(EXIT_FAILURE);
    }

    free(m1);
    free(m2);
    free(m3);

    return 0;
}
